
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void simpleMerge(int *items, int *freqs, int *result, int *pos) {

	int idx = threadIdx.x;

	printf("ThreadId: %d", idx);

	int item = items[idx];
	int freq = freqs[idx];

	int position = pos[idx];

	printf( "Put %d at %d, %d times\n", item, position, freq );

	for (int i = 0; i < freq; i++) {
		result[position+i] = item;
	}

	__syncthreads();
}



int main(void) {

	int items[5] = { 2, 5, 8, 2, 10 };
	int freqs[5] = { 10, 3, 0, 6, 5 };
	
	int pos[5];

	int *deviceItems, *deviceFreqs, *result, *devicePos;

	int resultSize = 0;

	for (int i = 0; i < 5; i++) {
		resultSize += freqs[i];
	}

	printf("%d\n", resultSize);
	fflush( stdout );

	int hostResult[resultSize];

	int curPos = 0;
	for (int i = 0; i < 5; i++) {

		if (i == 0) {
			pos[i] = 0;
		}
		else {
			curPos += freqs[i-1];
			pos[i] = curPos;
		}

	}

	printf("%d\n", curPos);

	hipMalloc( (void **)&deviceItems, 5 * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, 5 * sizeof(int));
	hipMalloc( (void **)&result, resultSize * sizeof(int));
	hipMalloc( (void **)&devicePos, 5 * sizeof(int));
	

	hipMemcpy( deviceItems, items, 5 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, 5 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos,   pos, 5 * sizeof(int), hipMemcpyHostToDevice);

	simpleMerge<<<1,5>>>(deviceItems, deviceFreqs, result, devicePos);

	hipMemcpy( hostResult, result, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	for (int i = 0; i < resultSize; i++) {
		printf("%d, ", hostResult[i]);
	}
}
