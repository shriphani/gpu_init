#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

using namespace mgpu;

__global__
void simpleMerge(int *items, int *freqs, int *result, int *pos, int numItems) {

	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	if (gid > numItems) {
		return;
	}

	printf("ThreadId: %d", gid);

	int item = items[gid];
	int freq = freqs[gid];

	int position = pos[gid];

	printf( "Put %d at %d, %d times\n", item, position, freq );

	for (int i = 0; i < freq; i++) {
		result[position+i] = item;
	}
}

int main(int argc, char ** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	int N = 5;

	int items[5] = { 2, 5, 8, 2, 10 };
	int freqs[5] = { 10, 3, 0, 6, 5 };

	int CTASize = 1024;
	int numBlocks = 1 + (N / CTASize);

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos, resultSize;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	simpleMerge<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N);

	int result[resultSize];

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	for (int i = 0; i < resultSize; i++) {
		printf("%d, ", result[i]);
	}

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);
}
