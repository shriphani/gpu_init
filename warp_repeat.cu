#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

#define CTA_SIZE 256

using namespace mgpu;

__global__
void warpRepeat(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

	const int nWarpsPerCTA = CTA_SIZE / 32;
	const int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const int wid = tid / 32;
	const int lane = tid % 32;
	const int lwid = wid % nWarpsPerCTA; 
	// tid 0 fetches the item and the frequency list
	int itemIdx = wid;

	if (itemIdx >= numItems)
		return;

	__shared__ int warpItem[CTA_SIZE / 32];
	__shared__ int warpFreq[CTA_SIZE / 32];

	if (lane == 0) {

		warpItem[lwid] = items[itemIdx];
		warpFreq[lwid] = freqs[itemIdx];

		//printf("Item it: %d\n, Item freq: %d\n", itemIdx, warpFreq);
	}

	__syncthreads();

	for (int i = lane; i < warpFreq[lwid]; i += 32) {
		//printf("item id: %d, freq: %d\n", lwid, warpFreq[lwid]);
		int position = pos[itemIdx] + i;
		result[position] = warpItem[lwid];
	}
}


int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {

	const int CTASize = 256;

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = (N * 32 + (CTASize - 1)) / CTA_SIZE;

	printf("Num blocks: %d\n", numBlocks);

	int *result = new int[resultSize];

	warpRepeat<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);	


	return result;
}