#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

using namespace mgpu;

__global__
void warpRepeat(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

	// tid 0 fetches the item and the frequency list
	int itemIdx = blockIdx.x;

	__shared__ int warpItem;
	__shared__ int warpFreq;

	if (threadIdx.x == 0) {

		warpItem = items[itemIdx];
		warpFreq = freqs[itemIdx];

		printf("Block id: %d\n", itemIdx);
	}

	__syncthreads();

	for (int i = threadIdx.x; i < warpFreq; i += 32) {
		int position = pos[itemIdx] + i;
		result[position] = warpItem;
	}
}


int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {

	int CTASize = 32;

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = N;

	printf("Num blocks: %d\n", numBlocks);

	int *result = new int[resultSize];

	warpRepeat<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);	


	return result;
}