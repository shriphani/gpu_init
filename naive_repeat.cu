#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

using namespace mgpu;

__global__
void simpleMerge(int *items, int *freqs, int *result, int *pos, int numItems) {

	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	if (gid > numItems) {
		return;
	}

	//printf("ThreadId: %d", gid);

	int item = items[gid];
	int freq = freqs[gid];

	int position = pos[gid];

	//printf( "Put %d at %d, %d times\n", item, position, freq );

	for (int i = 0; i < freq; i++) {
		result[position+i] = item;
	}
}

int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {
	int CTASize = 1024;
	int numBlocks = 1 + (N / CTASize);

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

#ifdef PROFILING
	hipProfilerStart();
#endif

	simpleMerge<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N);

#ifdef PROFILING
	hipProfilerStop();
#endif


	int *result = new int[resultSize];

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);

	return result;
}
