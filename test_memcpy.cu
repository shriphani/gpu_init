
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void cpTest(int *x) {
	int idx = threadIdx.x;

	printf( "At id: %d, Val: %d", idx, x[idx] );


}

int main(void) {

	int foo[5] = { 1, 2, 3, 4, 5 };

	int *deviceFoo;

	hipMalloc( (void **)&deviceFoo, 5 * sizeof(int) );
	hipMemcpy( deviceFoo, foo, 5 * sizeof(int), hipMemcpyHostToDevice );

	cpTest<<<1, 5>>>(deviceFoo);
	fflush(stdout);

}
