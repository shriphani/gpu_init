#include "hip/hip_runtime.h"
#include "moderngpu.cuh"

using namespace mgpu;

void scanReduceShit(CudaContext &context) {
	int N = 1000;

	MGPU_MEM(int) data = context.GenRandom<int>(N, 0, 9);
	printf("Input Array\n");
	PrintArray(*data, "%4d", 10);

	// reduce - sum
	int total = Reduce(data->get(), N, context);
	printf("Total: %d\n", total);

	// reduce - max
	int reduce;
	Reduce(data->get(), N, INT_MIN, maximum<int>(), (int*)0, &reduce, context);
	printf("Max: %d", reduce);
}

int main(int argc, char** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	scanReduceShit(*context);

    return 0;
}