#include "moderngpu.cuh"

using namespace mgpu;

void scanReduceShit(CudaContext &context) {
	int N = 1000;

	int data2[N];

	for (int i = 0; i < N; i++) {
		data2[i] = i;
	}

	int *deviceData2;

	hipMalloc((void **) &deviceData2, sizeof(int) * N);
	hipMemcpy((void *) deviceData2, (const void *) data2, sizeof(int) * N, hipMemcpyHostToDevice);

	// reduce - sum
	int total = Reduce(deviceData2, N, context);
	printf("Total: %d\n", total);

	ScanExc(deviceData2, N, &total, context);
	printf("Total: %d\n", total);

}

int main(int argc, char** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	scanReduceShit(*context);

    return 0;
}