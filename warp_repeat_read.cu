#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"
#include <math.h>

#define CTA_SIZE 256
#define WARP_SIZE 32

using namespace mgpu;

__global__
void warpRepeat2(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

    const int nWarpsPerCTA = CTA_SIZE / WARP_SIZE;
    const int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    const int wid = tid / WARP_SIZE;
    const int lane = tid % WARP_SIZE;
    const int lwid = wid % nWarpsPerCTA;

    // one item per warp
    volatile __shared__ int warpItem[CTA_SIZE / WARP_SIZE];
    volatile __shared__ int warpFreq[CTA_SIZE / WARP_SIZE];
    volatile __shared__ int warpPos[CTA_SIZE / WARP_SIZE];

    if (threadIdx.x < CTA_SIZE / WARP_SIZE) {

        int itemId = nWarpsPerCTA * blockIdx.x + threadIdx.x;

        if (itemId < numItems) {
            warpItem[threadIdx.x] = items[itemId];
            warpFreq[threadIdx.x] = freqs[itemId];
            warpPos[threadIdx.x] = pos[itemId];
        }
    }

    __syncthreads();

    for (int i = lane; i < warpFreq[lwid]; i += 32) {
        //printf("Lane: %d, wid: %d, item %d, position %d\n", lane, wid, warpItem[lwid], warpPos[lwid]);
        int position = warpPos[lwid] + i;
        result[position] = warpItem[lwid];
    }

}

int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = (N * WARP_SIZE + (CTA_SIZE - 1)) / CTA_SIZE;

	printf("Num blocks: %d\n", numBlocks);

	int *result = new int[resultSize];

	warpRepeat2<<<numBlocks, CTA_SIZE>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);


	return result;
}
