#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

#define CTA_SIZE 256
#define WARP_SIZE 32

using namespace mgpu;

__global__
void warpRepeat3(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

    const int nWarpsPerCTA = CTA_SIZE / WARP_SIZE;
    const int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    const int wid = tid / WARP_SIZE;
    const int lane = tid % WARP_SIZE;
    const int lwid = wid % nWarpsPerCTA;

    volatile __shared__ int ctaItem[CTA_SIZE];
    volatile __shared__ int ctaFreq[CTA_SIZE];
    volatile __shared__ int ctaPos[CTA_SIZE];

    // populate the block-specific shared mem items

    if (tid < numItems) {

        ctaItem[threadIdx.x] = items[tid];
        ctaFreq[threadIdx.x] = freqs[tid];
        ctaPos[threadIdx.x] = pos[tid];
    }

    // thunk executed by a warp
    for (int idx = 0; idx < WARP_SIZE; idx++) { // loop over all elements written by this warp
        // current thread reads out whatever it wrote
        // to shared mem.
        int currentThreadItem = (lwid * WARP_SIZE) + idx;

        //if (idx == 0)
            //printf("Current warp: %d, current idx: %d, item: %d, freq: %d\n", lwid, idx, ctaItem[currentThreadItem], ctaFreq[currentThreadItem]);

        for (int i = lane; i < ctaFreq[currentThreadItem]; i++) {
            int position = ctaPos[currentThreadItem] + i;
            //printf("Position: %d\n", position);
            result[position] = ctaItem[currentThreadItem];
        }

    }
}


int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = (N * WARP_SIZE + (CTA_SIZE - 1)) / CTA_SIZE;

	printf("Num blocks: %d\n", numBlocks);

	int *result = new int[resultSize];

	warpRepeat3<<<numBlocks, CTA_SIZE>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);


	return result;
}
