#include "hip/hip_runtime.h"
#include "repeat.cuh"


using namespace mgpu;

/**
 * 
 * Repeat based on binary search
 * Build a positions list (using a scan)
 * Work is divided based on the final results list
 * What idiot to plug in where is decided based on a binary search in
 * the positions list
 **/
__global__
void binarySearchRepeat(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

	// writing to
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);

	if (gid >= numResults) {
		return;
	}

	// what to write
	int itemIdx = BinarySearch<MgpuBoundsLower>(pos, numItems, gid, less_equal<int>());

	result[gid] = items[itemIdx - 1]; 

}

/**
 * Set parts up and so on.
 **/
int *partitionAndRun(int *items, int *freqs, int N, int &resultSize, ContextPtr context) {

	int CTASize = 1024;

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = 1 + (resultSize / CTASize);

	int *result = new int[resultSize];

#ifdef PROFILING
	printf("PROFILING!!!\n");
	hipProfilerStart();
#endif

	binarySearchRepeat<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

#ifdef PROFILING
	hipProfilerStop();
#endif

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);	


	return result;
}
