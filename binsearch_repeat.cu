#include "hip/hip_runtime.h"
#include <stdio.h>
#include "moderngpu.cuh"

using namespace mgpu;

/**
 * 
 * Repeat based on binary search
 * Build a positions list (using a scan)
 * Work is divided based on the final results list
 * What idiot to plug in where is decided based on a binary search in
 * the positions list
 **/
__global__
void binarySearchRepeat(int *items, int *freqs, int *result, int *pos, int numItems, int numResults) {

	// writing to
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);

	if (gid >= numResults) {
		return;
	}

	// what to write
	int itemIdx = BinarySearch<MgpuBoundsLower>(pos, numItems, gid, less_equal<int>());

	printf("At %d, writing %d\n", gid, itemIdx - 1);

	result[gid] = items[itemIdx - 1]; 

}

int main(int argc, char ** argv) {
	ContextPtr context = CreateCudaDevice(argc, argv, true);

	int N = 5;

	int items[5] = { 2, 5, 8, 2, 10 };
	int freqs[5] = { 10, 3, 0, 6, 5 };

	int CTASize = 1024;

	int *deviceItems, *deviceFreqs, *deviceResult, *devicePos, resultSize;

	hipMalloc( (void **)&deviceItems, N * sizeof(int));
	hipMalloc( (void **)&deviceFreqs, N * sizeof(int));
	hipMalloc( (void **)&devicePos, N * sizeof(int));

	hipMemcpy( deviceItems, items, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( deviceFreqs, freqs, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( devicePos, freqs, N * sizeof(int), hipMemcpyHostToDevice);

	ScanExc(devicePos, N, &resultSize, *context);

	hipMalloc( (void **)&deviceResult, resultSize * sizeof(int));

	int numBlocks = 1 + (resultSize / CTASize);

	binarySearchRepeat<<<numBlocks, CTASize>>>(deviceItems, deviceFreqs, deviceResult, devicePos, N, resultSize);

	int result[resultSize];

	hipMemcpy( result, deviceResult, resultSize * sizeof(int), hipMemcpyDeviceToHost );

	for (int i = 0; i < resultSize; i++) {
		printf("%d, ", result[i]);
	}

	hipFree(deviceItems);
	hipFree(deviceFreqs);
	hipFree(deviceResult);
	hipFree(devicePos);
}
